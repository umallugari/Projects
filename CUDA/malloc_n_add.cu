
#include <stdio.h>
#include <hip/hip_runtime.h>

//#include <cudaMalloc.h>

__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main(void)
{
    int a, b, c;
    int *pa, *pb, *pc;

    int size = sizeof(int);

    hipMalloc((void **)&pa, size);
    hipMalloc((void **)&pb, size);
    hipMalloc((void **)&pc, size);

    a = 7;
    b = 8;

    hipMemcpy(pa, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(pb, &b, size, hipMemcpyHostToDevice);

    add<<<1,1>>>(pa, pb, pc);

    hipMemcpy(&c, pc, size, hipMemcpyDeviceToHost);
    hipFree(pa);
    hipFree(pb);
    hipFree(pc);

    printf("GPU computed value of c (a+b): %d\n", c);

    return 0;
}
